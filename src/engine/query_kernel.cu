#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/remove.h>

#include <cstdlib>

#include "engine/query_kernel.h"
#include "engine/search_engine_gpu.hpp"

__device__ bool is_lyric_in_song(const char *lyric, const char *song) {
  int i = 0;
  int j = 0;

  while (lyric[i] != '\0' && song[j] != '\0') {
    if (lyric[i] == song[j]) {
      i++;
      j++;
    } else {
      i = 0;
      j++;
    }
  }

  if (lyric[i] == '\0') {
    return true;
  } else {
    // reached end of song without finding full lyric
    return false;
  }
}

__global__ void query_kernel(const char *lyric_to_search, const char *d_lyrics,
                             const int *d_song_indices, int num_songs,
                             int *d_results) {
  int song_index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = song_index; i < num_songs; i += stride) {
    if (is_lyric_in_song(lyric_to_search, d_lyrics + d_song_indices[i])) {
      d_results[i] = i;
    }
  }
}

void launch_query_kernel(const char *lyric_to_search, int lyric_len,
                         const char *d_lyrics, const int *d_song_indices,
                         int num_songs, int **h_results, int *num_results,
                         float *kernel_time_ms, float *data_transfer_time_ms) {
  char *d_lyric_to_search;

  float total_data_transfer_time_ms = 0.0, dt_ms = 0.0;
  hipEvent_t data_transfer_start, data_transfer_stop;
  hipEventCreate(&data_transfer_start);
  hipEventCreate(&data_transfer_stop);

  hipEventRecord(data_transfer_start);
  gpuErrchk(
      hipMalloc((void **)&d_lyric_to_search, (lyric_len + 1) * sizeof(char)));
  gpuErrchk(hipMemcpy(d_lyric_to_search, lyric_to_search,
                       (lyric_len + 1) * sizeof(char), hipMemcpyHostToDevice));
  hipEventRecord(data_transfer_stop);
  gpuErrchk(hipDeviceSynchronize());
  hipEventElapsedTime(&dt_ms, data_transfer_start, data_transfer_stop);
  total_data_transfer_time_ms += dt_ms;

  thrust::device_vector<int> d_results(num_songs, -1);

  int threads_per_block = 128;
  int num_blocks = (num_songs + threads_per_block - 1) / threads_per_block;

  hipEvent_t kernel_start, kernel_stop;
  hipEventCreate(&kernel_start);
  hipEventCreate(&kernel_stop);

  hipEventRecord(kernel_start);
  query_kernel<<<num_blocks, threads_per_block>>>(
      d_lyric_to_search, d_lyrics, d_song_indices, num_songs,
      thrust::raw_pointer_cast(d_results.data()));
  hipEventRecord(kernel_stop);
  gpuErrchk(hipDeviceSynchronize());
  hipEventElapsedTime(kernel_time_ms, kernel_start, kernel_stop);

  // reduce results to only valid indices
  // using gpu
  auto new_end = thrust::remove(d_results.begin(), d_results.end(), -1);
  d_results.erase(new_end, d_results.end());

  hipEventRecord(data_transfer_start);
  *h_results = (int *)malloc(d_results.size() * sizeof(int));
  gpuErrchk(hipMemcpy(*h_results, thrust::raw_pointer_cast(d_results.data()),
                       d_results.size() * sizeof(int), hipMemcpyDeviceToHost));
  *num_results = d_results.size();
  hipEventRecord(data_transfer_stop);
  gpuErrchk(hipDeviceSynchronize());
  hipEventElapsedTime(&dt_ms, data_transfer_start, data_transfer_stop);
  total_data_transfer_time_ms += dt_ms;
  *data_transfer_time_ms = total_data_transfer_time_ms;

  // cleanup
  gpuErrchk(hipFree(d_lyric_to_search));
  hipEventDestroy(data_transfer_start);
  hipEventDestroy(data_transfer_stop);
  hipEventDestroy(kernel_start);
  hipEventDestroy(kernel_stop);
}